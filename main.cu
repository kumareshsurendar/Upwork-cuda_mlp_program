#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "coordinates_dataset.h"
#include "bce_cost.h"
#include "matrix.h"

// Forward declarations of kernels
__global__ void reluForward(float *input, float *output, int size);
__global__ void sigmoidBackward(float *output, float *gradOutput, float *gradInput, int size);
__global__ void linearForward(float *input, float *weights, float *bias, float *output, int inputSize, int outputSize);


float computeAccuracy(float* predictions, float* targets, int count) {
    int correct = 0;
    for (int i = 0; i < count; ++i) {
        int pred_label = predictions[i] >= 0.5f ? 1 : 0;
        if (pred_label == static_cast<int>(targets[i])) {
            correct++;
        }
    }
    return static_cast<float>(correct) / count;
}

int main() {
    srand(time(NULL));

    const int samples = 1024;
    const int inputSize = 2;
    const int hiddenSize = 4;
    const int outputSize = 1;

    CoordinatesDataset dataset(samples, 1);
    Matrix& inputMatrix = dataset.getBatches()[0];
    Matrix& targetMatrix = dataset.getTargets()[0];
    inputMatrix.copyDeviceToHost();
    targetMatrix.copyDeviceToHost();

    float* hostInput = inputMatrix.data_host.get();
    float* hostLabels = targetMatrix.data_host.get();

    float *d_input, *d_hidden, *d_output, *d_weights1, *d_bias1, *d_weights2, *d_bias2;
    hipMalloc(&d_input, samples * inputSize * sizeof(float));
    hipMalloc(&d_hidden, samples * hiddenSize * sizeof(float));
    hipMalloc(&d_output, samples * outputSize * sizeof(float));
    hipMalloc(&d_weights1, hiddenSize * inputSize * sizeof(float));
    hipMalloc(&d_bias1, hiddenSize * sizeof(float));
    hipMalloc(&d_weights2, outputSize * hiddenSize * sizeof(float));
    hipMalloc(&d_bias2, outputSize * sizeof(float));

    hipMemset(d_weights1, 0, hiddenSize * inputSize * sizeof(float));
    hipMemset(d_bias1, 0, hiddenSize * sizeof(float));
    hipMemset(d_weights2, 0, outputSize * hiddenSize * sizeof(float));
    hipMemset(d_bias2, 0, outputSize * sizeof(float));

    hipMemcpy(d_input, hostInput, samples * inputSize * sizeof(float), hipMemcpyHostToDevice);

    for (int i = 0; i < samples; ++i) {
        float* sampleInput = d_input + i * inputSize;
        float* sampleHidden = d_hidden + i * hiddenSize;
        float* sampleOutput = d_output + i * outputSize;

        linearForward<<<1, hiddenSize>>>(sampleInput, d_weights1, d_bias1, sampleHidden, inputSize, hiddenSize);
        reluForward<<<1, hiddenSize>>>(sampleHidden, sampleHidden, hiddenSize);
        linearForward<<<1, outputSize>>>(sampleHidden, d_weights2, d_bias2, sampleOutput, hiddenSize, outputSize);
    }

    std::vector<float> predictions(samples);
    hipMemcpy(predictions.data(), d_output, samples * sizeof(float), hipMemcpyDeviceToHost);

    float accuracy = computeAccuracy(predictions.data(), hostLabels, samples);
    std::cout << "Network accuracy: " << accuracy << std::endl;

    hipFree(d_input);
    hipFree(d_hidden);
    hipFree(d_output);
    hipFree(d_weights1);
    hipFree(d_bias1);
    hipFree(d_weights2);
    hipFree(d_bias2);

    return 0;
}
